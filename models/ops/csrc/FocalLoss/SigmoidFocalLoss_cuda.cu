#include "hip/hip_runtime.h"
// modified from
// https://github.com/pytorch/pytorch/blob/master/modules/detectron/sigmoid_focal_loss_op.cu

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <cfloat>

// TODO make it in a common file
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)

template <typename T>
__global__ void SigmoidFocalLossForward(
    const int nthreads,
    const T* logits,
    const int64_t* targets,
    const int num_classes,
    const T gamma,
    const T alpha,
    const int num,
    T* losses) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    int n = i / num_classes;
    int d = i % num_classes; // current class[0~79];
    int64_t t = targets[n]; // target class [0~79];

    // Decide it is positive or negative case.
    T c1 = (t == d);
    T c2 = (t >= 0 & t != d);

    // p = 1. / 1. + expf(-x); p = sigmoid(x)
    T  p = (T)1. / ((T)1. + exp(-logits[i]));

    // (1-p)**gamma * log(p) where
    T term1 = pow(((T)1. - p), gamma) * log(max(p, (T)FLT_MIN));

    // p**gamma * log(1-p)
    // T term2 = pow(p, gamma) * log(max((T)1. - p, (T)FLT_MIN));
    T term2 = pow(p, gamma) *
        ((T)-1. * logits[i] * (logits[i] >= (T)0.) -
         log((T)1. + exp(logits[i] - (T)2. * logits[i] * (logits[i] >= (T)0.))));

    losses[i] = (T)0.;
    losses[i] += -c1 * term1 * alpha;
    losses[i] += -c2 * term2 * ((T)1. - alpha);
  } // CUDA_1D_KERNEL_LOOP
} // SigmoidFocalLossForward


template <typename T>
__global__ void SigmoidFocalLossBackward(
    const int nthreads,
    const T* logits,
    const int64_t* targets,
    const T* d_losses,
    const int num_classes,
    const T gamma,
    const T alpha,
    const int num,
    T* d_logits) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {

    int n = i / num_classes;
    int d = i % num_classes; // current class[0~79];
    int64_t t = targets[n]; // target class [0~79], 80 is background;

    // Decide it is positive or negative case.
    T c1 = (t == d);
    T c2 = (t >= 0 & t != d);

    // p = 1. / 1. + exp(-x); p = sigmoid(x)
    T  p = (T)1. / ((T)1. + exp(-logits[i]));

    // (1-p)**g * (1 - p - g*p*log(p)
    T term1 = pow(((T)1. - p), gamma) *
        ((T)1. - p - (p * gamma * log(max(p, (T)FLT_MIN))));

    // (p**g) * (g*(1-p)*log(1-p) - p)
    // T term_n = pow(p, gamma) *
    // (gamma * ((T)1. - p) * log(max((T)1. - p, (T)FLT_MIN)) - p);
    T term2 = pow(p, gamma) *
        (((T)-1. * logits[i] * (logits[i] >= (T)0.) -
         log((T)1. + exp(logits[i] - (T)2. * logits[i] * (logits[i] >= (T)0.)))) *
         ((T)1. - p) * gamma - p);

    d_logits[i] = (T)0.;
    d_logits[i] += -c1 * term1 * alpha;
    d_logits[i] += -c2 * term2 * ((T)1. - alpha);
    d_logits[i] = d_logits[i] * d_losses[i];
  } // CUDA_1D_KERNEL_LOOP
} // SigmoidFocalLossBackward

namespace pet {

at::Tensor SigmoidFocalLoss_forward_cuda(
		const at::Tensor& logits,
    const at::Tensor& targets,
		const int num_classes,
		const float gamma,
		const float alpha) {
  AT_ASSERTM(logits.device().is_cuda(), "logits must be a CUDA tensor");
  AT_ASSERTM(targets.device().is_cuda(), "targets must be a CUDA tensor");
  AT_ASSERTM(logits.dim() == 2, "logits should be NxClass");

  const int num_samples = logits.size(0);
  auto losses = at::empty({num_samples, logits.size(1)}, logits.options());
  auto losses_size = num_samples * logits.size(1);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(
    at::cuda::ATenCeilDiv(
        static_cast<int64_t>(losses_size), static_cast<int64_t>(512)),
    static_cast<int64_t>(4096)));
  dim3 block(512);

  if (losses.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return losses;
  }

  AT_DISPATCH_FLOATING_TYPES(logits.scalar_type(), "SigmoidFocalLoss_forward", [&] {
    SigmoidFocalLossForward<scalar_t><<<grid, block, 0, stream>>>(
        losses_size,
        logits.contiguous().data_ptr<scalar_t>(),
	      targets.contiguous().data_ptr<int64_t>(),
        num_classes,
	      gamma,
	      alpha,
	      num_samples,
        losses.data_ptr<scalar_t>());
  });
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());
  return losses;
}

at::Tensor SigmoidFocalLoss_backward_cuda(
		const at::Tensor& logits,
    const at::Tensor& targets,
		const at::Tensor& d_losses,
		const int num_classes,
		const float gamma,
		const float alpha) {
  AT_ASSERTM(logits.device().is_cuda(), "logits must be a CUDA tensor");
  AT_ASSERTM(targets.device().is_cuda(), "targets must be a CUDA tensor");
  AT_ASSERTM(d_losses.device().is_cuda(), "d_losses must be a CUDA tensor");
  AT_ASSERTM(logits.dim() == 2, "logits should be NxClass");

  const int num_samples = logits.size(0);
  AT_ASSERTM(logits.size(1) == num_classes, "logits.size(1) should be num_classes");

  auto d_logits = at::zeros({num_samples, num_classes}, logits.options());
  auto d_logits_size = num_samples * logits.size(1);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(
    at::cuda::ATenCeilDiv(
        static_cast<int64_t>(d_logits_size), static_cast<int64_t>(512)),
    static_cast<int64_t>(4096)));
  dim3 block(512);

  if (d_logits.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return d_logits;
  }

  AT_DISPATCH_FLOATING_TYPES(logits.scalar_type(), "SigmoidFocalLoss_backward", [&] {
    SigmoidFocalLossBackward<scalar_t><<<grid, block, 0, stream>>>(
        d_logits_size,
        logits.contiguous().data_ptr<scalar_t>(),
	      targets.contiguous().data_ptr<int64_t>(),
	      d_losses.contiguous().data_ptr<scalar_t>(),
        num_classes,
	      gamma,
	      alpha,
	      num_samples,
        d_logits.data_ptr<scalar_t>());
  });
  AT_CUDA_CHECK(hipGetLastError());
  return d_logits;
}

} // namespace pet